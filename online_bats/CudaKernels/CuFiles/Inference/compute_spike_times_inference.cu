
#include <hip/hip_runtime.h>
#define INFINITY __int_as_float(0x7f800000)

extern "C" {
    __device__ void get_sample_params(const int **spike_indices,
                                      const float **spike_times,
                                      const float **exp_tau_s,
                                      const float **exp_tau,
                                      const float **weights,
                                      int n_post_neurons, int n_pre_neurons, int sample_idx, int neuron_idx,
                                      int max_n_pre_spike) {
        int offset = sample_idx * max_n_pre_spike;

        *spike_indices += offset;
        *spike_times += offset;
        *exp_tau_s += offset;
        *exp_tau += offset;

        *weights += neuron_idx * n_pre_neurons;
    }

    __device__ void get_neuron_results(int **n_spikes,
                                       float **spike_times,
                                       int n_pre_neurons,
                                       int n_post_neurons,
                                       int sample_idx,
                                       int neuron_idx,
                                       int max_n_post_spike) {
        int sample_neuron_idx = (sample_idx * n_post_neurons + neuron_idx);
        int offset = sample_neuron_idx * max_n_post_spike;

        *n_spikes += sample_neuron_idx;
        *spike_times += offset;
    }

    __device__ bool compute_spikes(const float c,
                                   int *n_spikes,
                                   float *spike_times,
                                   float a,
                                   float *b,
                                   float last_spike,
                                   float next_spike,
                                   float tau,
                                   float max_simulation,
                                   int neuron_idx,
                                   int n_pre_neurons,
                                   int max_n_post_spike,
                                   int sample_idx) {
        float x, inside_log, tmp;

        // Compute until there is no spike anymore
        while (true) {
            // Compute discriminant
            tmp = (*b) * (*b) - 4 * a * c;

            if (tmp < 0) // Negative discriminant, no spike --> stop
                return false;
            x = sqrtf(tmp);
            tmp = x + (*b);

            if (tmp == 0.0) // Division per zero, no spike --> stop
                return false;
            inside_log = 2.0 * a / tmp;

            if (inside_log <= 0.0) // Negative log, no spike --> stop
                return false;

            tmp = tau * __logf(inside_log);

            // Spike time is before the last pre-spike or after the next spike --> stop
            if (tmp < last_spike || tmp > max_simulation || tmp > next_spike)
                return false;

            // Spike time is valid
            spike_times[*n_spikes] = tmp;
            last_spike = tmp;

            *b = a / inside_log; // Apply reset to b
            (*n_spikes)++;
            if (*n_spikes >= max_n_post_spike) {
                return true;
            }
        }
    }

    __global__ void compute_spike_times_kernel(// Parameters
                                               const int *spike_indices,
                                               const float *spike_times,
                                               const float *exp_tau_s,
                                               const float *exp_tau,
                                               const float *weights,
                                               const float c,
                                               float tau,
                                               float max_simulation,
                                               int max_n_pre_spike,
                                               int max_n_post_spike,
                                               int n_pre_neurons,
                                               // Outputs
                                               int *n_spikes,
                                               float *out_spike_times) {
        int n_neurons = blockDim.x;
        int sample_idx = blockIdx.x;
        int neuron_idx = threadIdx.x;

        get_sample_params(&spike_indices, &spike_times, &exp_tau_s, &exp_tau, &weights,
                          n_neurons, n_pre_neurons, sample_idx, neuron_idx, max_n_pre_spike);
        get_neuron_results(&n_spikes, &out_spike_times, n_pre_neurons, n_neurons, sample_idx, neuron_idx,
                           max_n_post_spike);

        float a = 0.0;
        float b = 0.0;
        int pre_idx;
        float weight;
        int next_i;
        float next_spike;
        float tmp;

        for (int i = 0; i < max_n_pre_spike; i++) {
            if (spike_times[i] == INFINITY) // No spike anymore --> stop
                break;
            pre_idx = spike_indices[i];
            weight = weights[pre_idx];

            // Update a and traces
            a += weight * exp_tau_s[i];

            // Update b
            b += weight * exp_tau[i];

            next_i = i + 1;
            if (next_i < max_n_pre_spike)
                next_spike = spike_times[next_i];
            else
                next_spike = INFINITY;

            if (compute_spikes(c, n_spikes, out_spike_times, a, &b, spike_times[i], next_spike,
                               tau, max_simulation, neuron_idx, n_pre_neurons,
                               max_n_post_spike, sample_idx))
                break; // Buffer full
        }
    }
}